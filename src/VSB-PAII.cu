/*
 ============================================================================
 Name        : VSB-PAII.cu
 Author      : Dave
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

/*
 ============================================================================
 Name        : cuda1.cu
 Author      : david
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

__global__ void VectorAdd(int *A,int *B,int *C,int M)
{
	int i = threadIdx.x;
	if(i < M)
	{
		C[i] = A[i] + B[i];
	}
}
__global__ void VectorAddMN(int *A,int *B,int *C,int M,int N)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < M && j < N)
	{
		C[i * M + j] = A[i * M + j] + B[i * M + j];
	}
}

__global__ void VectorAddMN_2(int *A,int *B,int *C,int M,int N)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < M && j < N)
	{

		for (int i = 0; i < M * N; ++i) {
			int row = i / N;
			int col = i % N;

			C[row + col] = A[row + col] + B[row + col];
		}
	}
}
void CV1_1()
{
	const unsigned int M = 10;
		const unsigned int bytes = M * sizeof(int);


		//Host allocate
		int *A_vectorHost = (int*)malloc(bytes);
		int *B_vectorHost = (int*)malloc(bytes);
		int *C_vectorHost = (int*)malloc(bytes);


		//Allocate the DEVICE memory to be able to copy data from HOST.
		int *A_vectorDevice;
		hipMalloc(&A_vectorDevice, bytes);

		int *B_vectorDevice;
		hipMalloc(&B_vectorDevice, bytes);

		int *C_vectorDevice;
		hipMalloc(&C_vectorDevice, bytes);


		for(int i=0; i < M;i++)
		{
			A_vectorHost[i] = i;
			B_vectorHost[i] = i;
			C_vectorHost[i] = 0;
		}
		hipMemcpy(A_vectorDevice, A_vectorHost, bytes, hipMemcpyHostToDevice);
		hipMemcpy(B_vectorDevice, B_vectorHost, bytes, hipMemcpyHostToDevice);
		hipMemcpy(C_vectorDevice, C_vectorHost, bytes, hipMemcpyHostToDevice);

		VectorAdd<<<1,M >>>(A_vectorDevice, B_vectorDevice, C_vectorDevice,M);

		hipMemcpy(C_vectorHost, C_vectorDevice, bytes, hipMemcpyDeviceToHost);

		for (int i = 0; i < 10; i++)
			{
				printf("\n %d", C_vectorHost[i]);
			}

		free(A_vectorHost);
		free(B_vectorHost);
		free(C_vectorHost);

		A_vectorHost = NULL;
		B_vectorHost = NULL;
		C_vectorHost = NULL;

		hipFree(A_vectorDevice);
		hipFree(B_vectorDevice);
		hipFree(C_vectorDevice);
}

void CV1_2()
{
	const unsigned int M = 10;
    const unsigned int N = 10;

	//Host allocate
	int *A_Host = (int*)malloc(M*N*sizeof(int));
	int *B_Host = (int*)malloc(M*N*sizeof(int));
	int *C_Host = (int*)malloc(M*N*sizeof(int));

	for(int i=0; i < M;i++)
	{
		for(int j=0; j < N;j++)
			{
			A_Host[i * M + j] = i  * j ;
			B_Host[i * M + j] = i  * j ;

			}
	}

	int *A_Device;
	hipMalloc(&A_Device,M*N*sizeof(int));

	int *B_Device;
	hipMalloc(&B_Device,M*N*sizeof(int));
	int *C_Device;
	hipMalloc(&C_Device,M*N*sizeof(int) );

	hipMemcpy(A_Device, A_Host,M*N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(B_Device, B_Host, M*N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(C_Device, C_Host, M*sizeof(int), hipMemcpyHostToDevice);

	dim3 dimBlock(M, N);
	dim3 dimGrid(1, 1);

	VectorAddMN<<<dimGrid, dimBlock>>>(A_Device,B_Device,C_Device,M,N);

	hipMemcpy(C_Host, C_Device, M*N*sizeof(int), hipMemcpyDeviceToHost);


	for (int i = 0; i < M; i++)
		{
			std::cout << i << "| " ;
			for (int j = 0; j < N; j++)
			{
				std::cout << C_Host[i * M + j] << "   ";
			}
			std::cout << std::endl;
		}

	free(A_Host);
	free(B_Host);
	free(C_Host);
	hipFree(A_Device);
	hipFree(B_Device);
	hipFree(C_Device);
};


void CV1_2_2()
{
	const unsigned int M = 10;
    const unsigned int N = 10;

    // M rows N cols
    int** A_Host = new int*[M];
    int** B_Host = new int*[M];
    int** C_Host = new int*[M];


    A_Host[0] = new int[M * N];
    B_Host[0] = new int[M * N];
    C_Host[0] = new int[M * N];

    for (int i = 1; i < M; ++i){
    	A_Host[i] = A_Host[i-1] + N;
    	B_Host[i] = B_Host[i-1] + N;
    	C_Host[i] = C_Host[i-1] + N;
    }

    //Fill
    for (int i = 0; i < M; ++i) {
     for (int j = 0; j < N; ++j) {
      A_Host[i][j] = i*j;
      B_Host[i][j] = i*j;
     }
    }


    int *A_Device;
    hipMalloc((void **)&A_Device, sizeof(int) * M * N);


    int *B_Device;
    hipMalloc((void **)&B_Device, sizeof(int) * M * N);


    int *C_Device;
    hipMalloc((void **)&C_Device, sizeof(int) * M * N);

    hipMemcpy(A_Device, A_Host[0],M*N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(B_Device, B_Host[0], M*N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(C_Device, C_Host[0], M*N*sizeof(int), hipMemcpyHostToDevice);

	dim3 dimBlock(M, N);
	dim3 dimGrid(1, 1);

	VectorAddMN_2<<<dimGrid, dimBlock>>>(A_Device,B_Device,C_Device,M,N);

	hipMemcpy(C_Host, C_Device, M*N*sizeof(int), hipMemcpyDeviceToHost);


	for (int i = 0; i < M; i++)
		{
			std::cout << i << "| " ;
			for (int j = 0; j < N; j++)
			{
				std::cout << C_Host[i][j] << "   ";
			}
			std::cout << std::endl;
		}

	free(A_Host);
	free(B_Host);
	free(C_Host);
	hipFree(A_Device);
	hipFree(B_Device);
	hipFree(C_Device);
};

int main(int argc, char *argv[])
{

CV1_2();

}



