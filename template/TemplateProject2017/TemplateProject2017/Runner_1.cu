
#include <hip/hip_runtime.h>
//#include <cudaDefs.h>
//
//cudaError_t error = cudaSuccess;
//cudaDeviceProp deviceProp = cudaDeviceProp();
//
///*
// ============================================================================
// Name        : VSB-PAII.cu
// Author      : Dave
// Version     :
// Copyright   : Your copyright notice
// Description : CUDA compute reciprocals
// ============================================================================
// */
//
// /*
//  ============================================================================
//  Name        : cuda1.cu
//  Author      : david
//  Version     :
//  Copyright   : Your copyright notice
//  Description : CUDA compute reciprocals
//  ============================================================================
//  */
//
//
//#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)
//
//constexpr unsigned int THREAD_PER_BLOCK = 256;
//constexpr unsigned int MEMBLOCK = 2;
//
//__global__ void VectorAdd(int* __restrict__ A, int* __restrict__ B, int* __restrict__ C, int M)
//{
//	unsigned int offset = threadIdx.x + blockIdx.x *  THREAD_PER_BLOCK;
//	unsigned int skip = gridDim.x * THREAD_PER_BLOCK;
//
//	while (offset < M)
//	{
//		C[offset] = A[offset] + B[offset];
//		offset += skip;
//	}
//}
//__global__ void VectorAddMN(int* __restrict__ A, int* __restrict__ B, int* __restrict__ C, int M, int N)
//{
//	const int i = blockIdx.x * blockDim.x + threadIdx.x;
//	const int j = blockIdx.y * blockDim.y + threadIdx.y;
//	if (i < M && j < N)
//	{
//		C[i * M + j] = A[i * M + j] + B[i * M + j];
//	}
//}
//
//__global__ void VectorAddMN_2(int *A, int *B, int *C, int M, int N)
//{
//	const int i = blockIdx.x * blockDim.x + threadIdx.x;
//	const int j = blockIdx.y * blockDim.y + threadIdx.y;
//	if (i < M && j < N)
//	{
//
//		for (int i = 0; i < M * N; ++i) {
//			int row = i / N;
//			int col = i % N;
//
//			C[row + col] = A[row + col] + B[row + col];
//		}
//	}
//}
//void CV1_1()
//{
//
//	//Time
//	cudaEvent_t startEvent, stopEvent;
//	float elapsedTime;
//	cudaEventCreate(&startEvent);
//	cudaEventCreate(&stopEvent);
//	cudaEventRecord(startEvent, 0);
//
//	
//	const unsigned int M = 10;
//	const unsigned int bytes = M * sizeof(int);
//
//	//Host allocate
//	int *A_vectorHost = (int*)malloc(bytes);
//	int *B_vectorHost = (int*)malloc(bytes);
//	int *C_vectorHost = (int*)malloc(bytes);
//
//
//	//Allocate the DEVICE memory to be able to copy data from HOST.
//	int *A_vectorDevice;
//	cudaMalloc((void**)&A_vectorDevice, bytes);
//
//	int *B_vectorDevice;
//	cudaMalloc((void**)&B_vectorDevice, bytes);
//
//	int *C_vectorDevice;
//	cudaMalloc((void**)&C_vectorDevice, bytes);
//
//
//	for (int i = 0; i < M; i++)
//	{
//		A_vectorHost[i] = i;
//		B_vectorHost[i] = i;
//		C_vectorHost[i] = 0;
//	}
//
//	cudaMemcpy(A_vectorDevice, A_vectorHost, bytes, cudaMemcpyHostToDevice);
//	cudaMemcpy(B_vectorDevice, B_vectorHost, bytes, cudaMemcpyHostToDevice);
//	cudaMemcpy(C_vectorDevice, C_vectorHost, bytes, cudaMemcpyHostToDevice);
//
//	dim3 dimBlock(THREAD_PER_BLOCK, THREAD_PER_BLOCK);
//	dim3 dimGrid(1, MEMBLOCK);
//
//
//	VectorAdd <<< dimGrid, dimBlock >>> (A_vectorDevice, B_vectorDevice, C_vectorDevice, M);
//
//	
//
//
//	cudaMemcpy(C_vectorHost, C_vectorDevice, bytes, cudaMemcpyDeviceToHost);
//
//	for (int i = 0; i < 10; i++)
//	{
//		printf("\n %d", C_vectorHost[i]);
//	}
//
//	free(A_vectorHost);
//	free(B_vectorHost);
//	free(C_vectorHost);
//
//	A_vectorHost = NULL;
//	B_vectorHost = NULL;
//	C_vectorHost = NULL;
//
//	cudaFree(A_vectorDevice);
//	cudaFree(B_vectorDevice);
//	cudaFree(C_vectorDevice);
//
//
//	//Time
//	cudaEventRecord(stopEvent, 0);
//	cudaEventSynchronize(stopEvent);
//	cudaEventElapsedTime(&elapsedTime, startEvent, stopEvent);
//
//	printf("Time to get device properties: %f ms", elapsedTime);
//	cudaEventDestroy(startEvent);
//	cudaEventDestroy(stopEvent);
//}
//
//void CV1_2()
//{
//	const unsigned int M = 10;
//	const unsigned int N = 10;
//
//	//Host allocate
//	int *A_Host = (int*)malloc(M*N * sizeof(int));
//	int *B_Host = (int*)malloc(M*N * sizeof(int));
//	int *C_Host = (int*)malloc(M*N * sizeof(int));
//
//	for (int i = 0; i < M; i++)
//	{
//		for (int j = 0; j < N; j++)
//		{
//			A_Host[i * M + j] = i * j;
//			B_Host[i * M + j] = i * j;
//
//		}
//	}
//
//	int *A_Device;
//	cudaMalloc(&A_Device, M*N * sizeof(int));
//
//	int *B_Device;
//	cudaMalloc(&B_Device, M*N * sizeof(int));
//	int *C_Device;
//	cudaMalloc(&C_Device, M*N * sizeof(int));
//
//	cudaMemcpy(A_Device, A_Host, M*N * sizeof(int), cudaMemcpyHostToDevice);
//	cudaMemcpy(B_Device, B_Host, M*N * sizeof(int), cudaMemcpyHostToDevice);
//	cudaMemcpy(C_Device, C_Host, M * sizeof(int), cudaMemcpyHostToDevice);
//
//	dim3 dimBlock(M, N);
//	dim3 dimGrid(1, 1);
//
//	VectorAddMN << <dimGrid, dimBlock >> > (A_Device, B_Device, C_Device, M, N);
//
//	cudaMemcpy(C_Host, C_Device, M*N * sizeof(int), cudaMemcpyDeviceToHost);
//
//
//	for (int i = 0; i < M; i++)
//	{
//		std::cout << i << "| ";
//		for (int j = 0; j < N; j++)
//		{
//			std::cout << C_Host[i * M + j] << "   ";
//		}
//		std::cout << std::endl;
//	}
//
//	free(A_Host);
//	free(B_Host);
//	free(C_Host);
//	cudaFree(A_Device);
//	cudaFree(B_Device);
//	cudaFree(C_Device);
//};
//
//
//void CV1_2_2()
//{
//	const unsigned int M = 10;
//	const unsigned int N = 10;
//
//	// M rows N cols
//	int** A_Host = new int*[M];
//	int** B_Host = new int*[M];
//	int** C_Host = new int*[M];
//
//
//	A_Host[0] = new int[M * N];
//	B_Host[0] = new int[M * N];
//	C_Host[0] = new int[M * N];
//
//	for (int i = 1; i < M; ++i) {
//		A_Host[i] = A_Host[i - 1] + N;
//		B_Host[i] = B_Host[i - 1] + N;
//		C_Host[i] = C_Host[i - 1] + N;
//	}
//
//	//Fill
//	for (int i = 0; i < M; ++i) {
//		for (int j = 0; j < N; ++j) {
//			A_Host[i][j] = i * j;
//			B_Host[i][j] = i * j;
//		}
//	}
//
//
//	int *A_Device;
//	cudaMalloc((void **)&A_Device, sizeof(int) * M * N);
//
//
//	int *B_Device;
//	cudaMalloc((void **)&B_Device, sizeof(int) * M * N);
//
//
//	int *C_Device;
//	cudaMalloc((void **)&C_Device, sizeof(int) * M * N);
//
//	cudaMemcpy(A_Device, A_Host[0], M*N * sizeof(int), cudaMemcpyHostToDevice);
//	cudaMemcpy(B_Device, B_Host[0], M*N * sizeof(int), cudaMemcpyHostToDevice);
//	cudaMemcpy(C_Device, C_Host[0], M*N * sizeof(int), cudaMemcpyHostToDevice);
//
//	dim3 dimBlock(M, N);
//	dim3 dimGrid(1, 1);
//
//	VectorAddMN_2 <<< dimGrid, dimBlock >>> (A_Device, B_Device, C_Device, M, N);
//
//	cudaMemcpy(C_Host, C_Device, M*N * sizeof(int), cudaMemcpyDeviceToHost);
//
//
//	for (int i = 0; i < M; i++)
//	{
//		std::cout << i << "| ";
//		for (int j = 0; j < N; j++)
//		{
//			std::cout << C_Host[i][j] << "  ";
//		}
//		std::cout << std::endl;
//	}
//
//	free(A_Host);
//	free(B_Host);
//	free(C_Host);
//	cudaFree(A_Device);
//	cudaFree(B_Device);
//	cudaFree(C_Device);
//};
////
////int main(int argc, char *argv[])
////{
////
////	CV1_1();
////
////}
////
//
