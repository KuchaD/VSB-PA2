
#include <hip/hip_runtime.h>
//#include <cudaDefs.h>
//
//cudaError_t error = cudaSuccess;
//cudaDeviceProp deviceProp = cudaDeviceProp();
//
//const unsigned ROWS = 15;
//const unsigned COLS = 20;
//const unsigned BLOCKSIZE = 8;
//
//__global__ void Fill(int* __restrict__  matrix, int rows, int cols, size_t pitch)
//{
//	int row = blockIdx.x * BLOCKSIZE + threadIdx.x;
//	int col = blockIdx.y * BLOCKSIZE + threadIdx.y;
//	if (row >= rows || col >= cols)
//		return;
//
//	int index = row * pitch + col;
//	int value = col * rows + row;
//	
//	matrix[index] = value;
//}
//
//__global__ void increment(int*  __restrict__ matrix, int rows, int cols, size_t pitch)
//{
//	int row = blockIdx.x * BLOCKSIZE + threadIdx.x;
//	int col = blockIdx.y * BLOCKSIZE + threadIdx.y;
//	if (row >= rows || col >= cols)
//		return;
//
//	int index = row * pitch + col;
//	int value = col * rows + row;
//	matrix[index]++;
//}
// int main(int argc, char *argv[])
//{
//	int *dMatrix;
//	size_t pitchBytes = 0;
//	checkCudaErrors(cudaMallocPitch((void**)&dMatrix, &pitchBytes, COLS * sizeof(int), ROWS));
//	size_t pitch = pitchBytes / sizeof(int);
//	
//	dim3 Grid = dim3(getNumberOfParts(ROWS, BLOCKSIZE), getNumberOfParts(COLS, BLOCKSIZE));
//	dim3 BLOCK = dim3(BLOCKSIZE, BLOCKSIZE);
//
//	Fill <<< Grid, BLOCK >>> (dMatrix, ROWS, COLS, pitch);
//	checkDeviceMatrix(dMatrix, pitchBytes, ROWS, COLS, "%-3d ", "dMatrix");
//
//
//	
//	int *matrix = new int[pitch * ROWS];
//	checkCudaErrors(cudaMemcpy2D(matrix, pitchBytes, dMatrix, pitchBytes, COLS * sizeof(int), ROWS, cudaMemcpyKind::cudaMemcpyDeviceToHost));
//	checkHostMatrix(matrix, pitchBytes, ROWS, COLS, "%-3d ", "matrix");
//
//	increment <<< Grid, BLOCK >>> (dMatrix, ROWS, COLS, pitch);
//	checkDeviceMatrix(dMatrix, pitchBytes, ROWS, COLS, "%-3d ", "dMatrixIncrement");
//
//
//	delete[] matrix;
//	cudaFree(dMatrix);
//	return 0;
//}