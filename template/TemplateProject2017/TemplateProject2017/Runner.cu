
#include <hip/hip_runtime.h>
//#include <cudaDefs.h>
//#include <time.h>
//#include <math.h>
//#include <random>
//
//using namespace std;
//
//
//cudaError_t error = cudaSuccess;
//cudaDeviceProp deviceProp = cudaDeviceProp();
//
//
//__constant__  __device__  int hodnota;
//
//typedef struct { int x; int y;} Point;
//__constant__  __device__  Point dPoint;
//__constant__  __device__ int dPole[5];
//
//
//
//void Cv1()
//{
//	initializeCUDA(deviceProp);
//
//
//	int hA = 100;
//	int hB = 0;
//	cudaMemcpyToSymbol(static_cast<const void*>(&hodnota), static_cast<const void*>(&hA), sizeof(hodnota));
//	cudaMemcpyFromSymbol(static_cast<void*>(&hB), static_cast<const void*>(&hodnota), sizeof(hodnota));
//
//	cout << hB << endl;
//
//
//
//	Point hL;
//	hL.x = 1;
//	hL.y = 2;
//
//	Point hL2;
//	cudaMemcpyToSymbol(static_cast<const void*>(&dPoint), static_cast<const void*>(&hL), sizeof(hL));
//	cudaMemcpyFromSymbol(static_cast<void*>(&hL2), static_cast<const void*>(&dPoint), sizeof(hL));
//
//	cout << hL2.x << " " << hL2.y << endl;
//
//	int Pole[5] = { 1,2,3,4,5 };
//	int hPole2[5];
//	cudaMemcpyToSymbol(dPole, Pole, sizeof(Pole));
//	cudaMemcpyFromSymbol(hPole2, dPole, sizeof(Pole));
//
//	cout << hPole2[0] << " " << hPole2[1];
//}
//
//
//int main(int argc, char *argv[])
//{
//	initializeCUDA(deviceProp);
//
//	//Cv1();
//	Cv2();
//	system("pause");
//
//}
