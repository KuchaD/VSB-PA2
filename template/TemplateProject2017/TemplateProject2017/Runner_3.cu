
#include <hip/hip_runtime.h>
//#include <cudaDefs.h>
//#include <time.h>
//#include <math.h>
//#include <random>
//
////WARNING!!! Do not change TPB and NO_FORCES for this demo !!!
//constexpr unsigned int TPB = 128; //thred per block
//constexpr unsigned int NO_FORCES = 256;
//constexpr unsigned int NO_RAIN_DROPS = 1 << 20;
//
//constexpr unsigned int MEM_BLOCKS_PER_THREAD_BLOCK = 8;
//constexpr unsigned int THREAD_PER_BLOCK = 50;
//
//cudaError_t error = cudaSuccess;
//cudaDeviceProp deviceProp = cudaDeviceProp();
//
//using namespace std;
//
//float3 *createData(const unsigned int length)
//{
//	//TODO: Generate float3 vectors. You can use 'make_float3' method.
//	float3 *data = static_cast<float3*>(::operator new(sizeof(float3)*length));
///*
//	std::random_device rd;
//	std::mt19937 gen(rd());
//	std::uniform_int_distribution<float> dis(0.0f, 1.0f);
//*/
//	for (unsigned int  i = 0; i < length; i++)
//	{
///*
//		
//		data[i].x = dis(gen);
//		data[i].y = dis(gen);
//		data[i].z = dis(gen);
//	*/
//		data[i] = make_float3(1, 1, 1);
//
//	}
//	
//	return data;
//}
//
//void printData(const float3 *data, const unsigned int length)
//{
//	if (data == 0) return;
//	const float3 *ptr = data;
//	for (unsigned int i = 0; i<length; i++, ptr++)
//	{
//		printf("%5.2f %5.2f %5.2f ", ptr->x, ptr->y, ptr->z);
//	}
//}
//
//////////////////////////////////////////////////////////////////////////////////////////////////////
///// <summary>	Sums the forces to get the final one using parallel reduction. 
///// 		    WARNING!!! The method was written to meet input requirements of our example, i.e. 128 threads and 256 forces  </summary>
///// <param name="dForces">	  	The forces. </param>
///// <param name="noForces">   	The number of forces. </param>
///// <param name="dFinalForce">	[in,out] If non-null, the final force. </param>
//////////////////////////////////////////////////////////////////////////////////////////////////////
//__global__ void reduce(const float3 * __restrict__ dForces, const unsigned int noForces, float3* __restrict__ dFinalForce)
//{
//	__shared__ float3 sForces[TPB];					//SEE THE WARNING MESSAGE !!!
//	unsigned int tid = threadIdx.x;
//	unsigned int next = TPB;						//SEE THE WARNING MESSAGE !!!
//
//	
//
//	
//	float3* src = (float3*)&sForces[tid];
//	float3* src2 = (float3*)&dForces[tid + next];
//
//
//	*src = dForces[tid];
//	
//	
//		
//		src->x += src2->x;
//		src->y += src2->y;
//		src->z += src2->z;
//
//		__syncthreads();
//
//		next >>= 1; // 64
//		if (tid >= next) return;
//
//		src2 = src + next;
//
//		src->x += src2->x;
//		src->y += src2->y;
//		src->z += src2->z;
//
//		__syncthreads();
//
//		next >>= 1; // 32
//		if (tid >= next) return;
//
//		src2 = src + next;
//
//		src->x += src2->x;
//		src->y += src2->y;
//		src->z += src2->z;
//
//		
//		volatile float3 *vsrc = &sForces[tid];
//		volatile float3 *vsrc2 = vsrc + next;
//		
//		next >>= 1; // 16
//		if (tid >= next) return;
//
//		vsrc2 = vsrc + next;
//
//		vsrc->x += vsrc2->x;
//		vsrc->y += vsrc2->y;
//		vsrc->z += vsrc2->z;
//
//		next >>= 1; // 8
//		if (tid >= next) return;
//
//		vsrc2 = vsrc + next;
//
//		vsrc->x += vsrc2->x;
//		vsrc->y += vsrc2->y;
//		vsrc->z += vsrc2->z;
//
//		next >>= 1; // 4
//		if (tid >= next) return;
//
//		vsrc2 = vsrc + next;
//
//		vsrc->x += vsrc2->x;
//		vsrc->y += vsrc2->y;
//		vsrc->z += vsrc2->z;
//
//		next >>= 1; // 2
//		if (tid >= next) return;
//
//		vsrc2 = vsrc + next;
//
//		vsrc->x += vsrc2->x;
//		vsrc->y += vsrc2->y;
//		vsrc->z += vsrc2->z;
//
//		next >>= 1; // 1
//		if (tid >= next) return;
//
//		vsrc2 = vsrc + next;
//
//		vsrc->x += vsrc2->x;
//		vsrc->y += vsrc2->y;
//		vsrc->z += vsrc2->z;
//
//		if (tid == 0)
//		{
//			dFinalForce->x = vsrc->x;
//			dFinalForce->y = vsrc->y;
//			dFinalForce->z = vsrc->z; 	
//		}
//}
//
//////////////////////////////////////////////////////////////////////////////////////////////////////
///// <summary>	Adds the FinalForce to every Rain drops position. </summary>
///// <param name="dFinalForce">	The final force. </param>
///// <param name="noRainDrops">	The number of rain drops. </param>
///// <param name="dRainDrops"> 	[in,out] If non-null, the rain drops positions. </param>
//////////////////////////////////////////////////////////////////////////////////////////////////////
//__global__ void add(const float3* __restrict__ dFinalForce, const unsigned int noRainDrops, float3* __restrict__ dRainDrops)
//{
//	
//	unsigned int bid = blockIdx.x * MEM_BLOCKS_PER_THREAD_BLOCK + threadIdx.x;
//#pragma unroll MEM_BLOCKS_PER_THREAD_BLOCK
//	for (size_t i = 0; i < MEM_BLOCKS_PER_THREAD_BLOCK; i++)
//	{
//		auto tid = bid + i;
//		if (tid >= noRainDrops)
//			return;
//		dRainDrops[tid].x += dFinalForce->x;
//		dRainDrops[tid].y += dFinalForce->y;
//		dRainDrops[tid].z += dFinalForce->z;
//	}
//}
//
//
//int main(int argc, char *argv[])
//{
//	initializeCUDA(deviceProp);
//
//	cudaEvent_t startEvent, stopEvent;
//	float elapsedTime;
//
//	cudaEventCreate(&startEvent);
//	cudaEventCreate(&stopEvent);
//	cudaEventRecord(startEvent, 0);
//
//	float3 *hForces = createData(NO_FORCES);
//	float3 *hDrops = createData(NO_RAIN_DROPS);
//
//	float3 *dForces = nullptr;
//	float3 *dDrops = nullptr;
//	float3 *dFinalForce = nullptr;
//
//	error = cudaMalloc((void**)&dForces, NO_FORCES * sizeof(float3));
//	error = cudaMemcpy(dForces, hForces, NO_FORCES * sizeof(float3), cudaMemcpyHostToDevice);
//
//	error = cudaMalloc((void**)&dDrops, NO_RAIN_DROPS * sizeof(float3));
//	error = cudaMemcpy(dDrops, hDrops, NO_RAIN_DROPS * sizeof(float3), cudaMemcpyHostToDevice);
//
//	error = cudaMalloc((void**)&dFinalForce, sizeof(float3));
//
//	KernelSetting ksReduce;
//
//	ksReduce.dimGrid = dim3(1, 1,1);
//	ksReduce.dimBlock = dim3(TPB, 1, 1);
//
//
//	KernelSetting ksAdd;
//	ksAdd.dimGrid = dim3(TPB, 1,1);
//	ksAdd.dimBlock = dim3(getNumberOfParts(NO_RAIN_DROPS, TPB * MEM_BLOCKS_PER_THREAD_BLOCK), 1, 1);
//	
//	for (unsigned int i = 0; i<1000; i++)
//	{
//		reduce<<<ksReduce.dimGrid, ksReduce.dimBlock>>>(dForces, NO_FORCES, dFinalForce);
//		add<<<ksAdd.dimGrid, ksAdd.dimBlock>>>(dFinalForce, NO_RAIN_DROPS, dDrops);
//	}
//
//	checkDeviceMatrix<float>((float*)dFinalForce, sizeof(float3), 1, 3, "%5.2f ", "Final force");
//	checkDeviceMatrix<float>((float*)dDrops, sizeof(float3), NO_RAIN_DROPS, 3, "%5.2f ", "Final Rain Drops");
//
//	if (hForces)
//		free(hForces);
//	if (hDrops)
//		free(hDrops);
//
//	cudaFree(dForces);
//	cudaFree(dDrops);
//
//	cudaEventRecord(stopEvent, 0);
//	cudaEventSynchronize(stopEvent);
//
//	cudaEventElapsedTime(&elapsedTime, startEvent, stopEvent);
//	cudaEventDestroy(startEvent);
//	cudaEventDestroy(stopEvent);
//
//	printf("Time to get device properties: %f ms", elapsedTime);
//}
