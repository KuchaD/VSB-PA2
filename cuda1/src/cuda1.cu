/*
 ============================================================================
 Name        : cuda1.cu
 Author      : david
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

__global__ void VectorAdd(int *A,int *B,int *C,int M)
{
	int i = threadIdx.x;
	if(i < M)
	{
		C[i] = A[i] + B[i];
	}
}
__global__
void VectorAddMN(int **A,int **B,int *C,int M,int N)
{
	int i= threadIdx.x + blockIdx.x * blockDim.x;
	int j= threadIdx.x + blockIdx.y * blockDim.y;

	if(i < M)
	{
		if(j < N){

			C[i] += A[i][j] + B[i][j];

		}
	}
}
void CV1_1()
{
	const unsigned int M = 10;
		const unsigned int bytes = M * sizeof(int);


		//Host allocate
		int *A_vectorHost = (int*)malloc(bytes);
		int *B_vectorHost = (int*)malloc(bytes);
		int *C_vectorHost = (int*)malloc(bytes);


		//Allocate the DEVICE memory to be able to copy data from HOST.
		int *A_vectorDevice;
		hipMalloc(&A_vectorDevice, bytes);

		int *B_vectorDevice;
		hipMalloc(&B_vectorDevice, bytes);

		int *C_vectorDevice;
		hipMalloc(&C_vectorDevice, bytes);


		for(int i=0; i < M;i++)
		{
			A_vectorHost[i] = i;
			B_vectorHost[i] = i;
			C_vectorHost[i] = 0;
		}
		hipMemcpy(A_vectorDevice, A_vectorHost, bytes, hipMemcpyHostToDevice);
		hipMemcpy(B_vectorDevice, B_vectorHost, bytes, hipMemcpyHostToDevice);
		hipMemcpy(C_vectorDevice, C_vectorHost, bytes, hipMemcpyHostToDevice);

		VectorAdd<<<1,M >>>(A_vectorDevice, B_vectorDevice, C_vectorDevice,M);

		hipMemcpy(C_vectorHost, C_vectorDevice, bytes, hipMemcpyDeviceToHost);

		for (int i = 0; i < 10; i++)
			{
				printf("\n %d", C_vectorHost[i]);
			}

		free(A_vectorHost);
		free(B_vectorHost);
		free(C_vectorHost);

		A_vectorHost = NULL;
		B_vectorHost = NULL;
		C_vectorHost = NULL;

		hipFree(A_vectorDevice);
		hipFree(B_vectorDevice);
		hipFree(C_vectorDevice);
}

void CV1_2()
{
	const unsigned int M = 10;
    const unsigned int N = 10;

	//Host allocate
	int **A_Host = (int**)malloc(M*N*sizeof(int));
	int **B_Host = (int**)malloc(M*N*sizeof(int));
	int *C_Host = (int*)malloc(M*sizeof(int));

	for(int i=0; i < M;i++)
	{
		for(int j=0; j < N;j++)
			{
				A_Host[i][j] = i * j;
				B_Host[i][j] = i * j;

			}
		C_Host[i] = 0;
	}

	int **A_Device;
	hipMalloc(&A_Device,M*N*sizeof(int));

	int **B_Device;
	hipMalloc(&B_Device,M*N*sizeof(int));
	int *C_Device;
	hipMalloc(&C_Device,M*sizeof(int) );

	hipMemcpy(A_Device, A_Host,M*N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(B_Device, B_Host, M*N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(C_Device, C_Host, M*sizeof(int), hipMemcpyHostToDevice);

	int Thread = M*N;

	VectorAddMN<<<2,Thread>>>(A_Device,B_Device,C_Device,M,N);

	hipMemcpy(C_Host, C_Device, M*sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0; i < 10; i++)
	{
	 printf("\n N:[%d] - %d",i, C_Host[i]);
	}


	free(A_Host);
	free(B_Host);
	free(C_Host);
	hipFree(A_Device);
	hipFree(B_Device);
	hipFree(C_Device);
};
int main(int argc, char *argv[])
{

CV1_2();

}



